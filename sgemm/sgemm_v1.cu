/* apply matrix multiplication A*B = C,
 * A: M rows, K cols
 * B: K rows, N cols
 * C: M rows, N cols
 */
#include <hip/hip_runtime.h>


#include <hipblas.h>

#include <iostream>
#include <sstream>
#include <cmath>

#define M 1400
#define N 1200
#define K 1024


namespace err {
std::string PrevFile{};
int PrevLine{0};

/**
 * helper function to check for errors in CUDA calls
 * source: NVIDIA
 * */
//#define NDEBUG
void checkErr(const std::string &file, int line) {
#ifndef NDEBUG
    hipError_t Error = hipGetLastError();
    if (Error != hipSuccess) {
        std::stringstream stream;
        stream << '\n'
               << file << ", line " << line << ": " << hipGetErrorString(Error) << " (" << Error
               << ")\n";
        if (PrevLine > 0) {
            stream << "Previous CUDA call:" << '\n' << PrevFile << ", line " << PrevLine << '\n';
        }
        throw std::runtime_error(stream.str());
    }
    PrevFile = file;
    PrevLine = line;
#endif
}
} // namespace err

#define CHECK_ERR err::checkErr(__FILE__, __LINE__)

template<unsigned int TILE_SIZE>
__global__ void sgemm_1(float * __restrict__ A, float * __restrict__ B, float * __restrict__ C,
                    const int m, const int k, const int n){
    __shared__ float shr_A[TILE_SIZE][TILE_SIZE];
    __shared__ float shr_B[TILE_SIZE][TILE_SIZE];
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    float val = 0.0f;
    for(int phase = 0; phase < std::ceil(k / (float) blockDim.x); ++phase){
        
        shr_A[ty][tx] = 0.0f;
        shr_B[ty][tx] = 0.0f;
        if(row < m && phase * TILE_SIZE + tx < k){
            shr_A[ty][tx] = A[row * k + phase * TILE_SIZE + tx];
        }
        if(col < n && phase * TILE_SIZE + ty < k){
            shr_B[ty][tx] = B[(phase*TILE_SIZE+ty) * n + col];
        }
        __syncthreads();
        for(int i = 0; i < blockDim.x; i++){
            val += shr_A[ty][i] * shr_B[i][tx];
        }
        __syncthreads();
    }

    if(row < m && col < n){
        C[row * n + col] = val;
    }

}

void initialize(float *X, int rows, int cols, bool flag = true);
void compute_ref_C(float *A, float *B, float *C, int m, int k, int n);
void print_matrix(float *A, int rows, int cols);
bool checkResult(float *C, float *ref, int rows, int cols);



int main(){
    hipEvent_t startTimer, stopTimer;
    hipEventCreate(&startTimer);
    hipEventCreate(&stopTimer);

    float *A = (float*) malloc(M * K * sizeof(float));
    float *B = (float*) malloc(K * N * sizeof(float));
    float *C = (float*) malloc(M * N * sizeof(float));
    float *ref_C = (float*) malloc(M * N * sizeof(float));

    float *dev_A, *dev_B, *dev_C;
    hipMalloc(&dev_A, M * K * sizeof(float)); CHECK_ERR;
    hipMalloc(&dev_B, K * N * sizeof(float)); CHECK_ERR;
    hipMalloc(&dev_C, M * N * sizeof(float)); CHECK_ERR;

    initialize(A, M, K);
    initialize(B, K, N);
    initialize(C, M, N, false);
    initialize(ref_C, M, N, false);
    compute_ref_C(A, B, ref_C, M, K, N);
    // print_matrix(ref_C, M, N);


    hipMemcpy(dev_A, A, M * K * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
    hipMemcpy(dev_B, B, K * N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;
    hipMemcpy(dev_C, C, M * N * sizeof(float), hipMemcpyHostToDevice); CHECK_ERR;

    const int block_size = 32;
    const int gridDim_x = std::ceil((float)N / (float)block_size);
    const int gridDim_y = std::ceil((float)M / (float)block_size);

    dim3 GRID(gridDim_x, gridDim_y,1);
    dim3 BLOCK(block_size, block_size, 1);

    hipEventRecord(startTimer);
    sgemm_1<block_size> <<<GRID, BLOCK>>> (dev_A, dev_B, dev_C, M, K, N);

    hipEventRecord(stopTimer);
    hipEventSynchronize(stopTimer);
    CHECK_ERR;
    float mySgemmTime{};
    hipEventElapsedTime(&mySgemmTime, startTimer, stopTimer);
    CHECK_ERR;


    hipMemcpy(C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    if(checkResult(C, ref_C, M, N) == true){
        std::cout << "Answer correct!" << std::endl;
    }else{
        std::cout << "Answer wrong!" << std::endl;
    }

    std::cout << "mySgemmTime = " << mySgemmTime <<" ms"<< std::endl;
    double FLOP = (double)M * (double)N * (double)K * 2.0f;
    double FLOPs = FLOP / (double) mySgemmTime / (double) 1000000;
    std::cout << "FLOPs = " << FLOPs << "GFLOPs" << std::endl;


    // compare with Cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    float alpha = 1.0f, beta = 0.0f;
    int numRepeats = 200;
    hipEventRecord(startTimer);
    for (int i = 0; i < numRepeats; ++i){
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, dev_B, N, dev_A, K, &beta, dev_C, N);
    }
    hipEventRecord(stopTimer);
    hipEventSynchronize(stopTimer);
    float cublasGemmTime{};
    hipEventElapsedTime(&cublasGemmTime, startTimer, stopTimer);
    cublasGemmTime /= (float)numRepeats;
    CHECK_ERR;

    std::cout << "============================================"<<std::endl;
    hipMemcpy(C, dev_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    if(checkResult(C, ref_C, M, N) == true){
        std::cout << "Cublas Answer correct!" << std::endl;
    }else{
        std::cout << "Cublas Answer wrong!" << std::endl;
    }
    std::cout << "CublasTime = " << cublasGemmTime <<" ms"<< std::endl;
    double cublasFLOPs = FLOP / (double) cublasGemmTime / (double) 1000000;
    std::cout << "FLOPs = " << cublasFLOPs << "GFLOPs" << std::endl;

    hipblasDestroy(handle);


    hipEventDestroy(startTimer);
    hipEventDestroy(stopTimer);
    
    hipFree(dev_A); CHECK_ERR;
    hipFree(dev_B); CHECK_ERR;
    hipFree(dev_C); CHECK_ERR;

    return 0;

}

void initialize(float *X, int rows, int cols, bool flag){
    if(flag){
        for(int i = 0; i < rows; i++){
            for(int j = 0; j < cols; j++){
                X[cols * i + j] = 0.5*i;
            }
        }
    }else{
        for(int i = 0; i < rows; i++){
            for(int j = 0; j < cols; j++){
                X[cols * i + j] = 0.0f;
            }
        }
    }
}
void compute_ref_C(float *A, float *B, float *C, int m, int k, int n){
    for(int i = 0; i < m; i++){
        for(int l = 0; l < k; l++){
            for(int j = 0; j < n; j++){
                C[n * i + j] += A[k * i + l] * B[n * l + j];
            }
        }
    }
}

void print_matrix(float *A, int rows, int cols){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
            std::cout << A[cols * i + j] << ' ';
        }
        std::cout << std::endl;
    }
}

bool checkResult(float *C, float *ref, int rows, int cols){
    for(int i = 0; i < rows; i++){
        for(int j = 0; j < cols; j++){
            if(fabs(C[i * cols + j] - ref[i * cols + j]) > 1e-10){
                return false;
            }
        }
    }
    return true;
}
